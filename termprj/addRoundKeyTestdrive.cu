#include "hip/hip_runtime.h"
#include "addRoundKey.h"
__global__ void wrapper(uint8_t* block, uint8_t* rdk);

int main(){
// try copy 2d to dev using cudamemcpy, works
    uint8_t data[][Ncol] = {0x0f,0x0f,0x0f,0x0f, 0x0f,0x0f, 0x0f,0x0f, 0x0f,0x0f, 0x0f,0x0f, 0x0f,0x0f, 0x0f,0x0f  }; 
    uint8_t *d_data,*d_key;
    uint8_t* result;
    uint8_t key[][Ncol] = {0xff,0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 0xff  }; 
    size_t size = Nrow*Ncol*sizeof(uint8_t);
    result =( uint8_t*) malloc(size);
    hipMalloc(&d_data, size);
    hipMalloc(&d_key, size);
    hipMemcpy(d_data,data,size,hipMemcpyHostToDevice);
    hipMemcpy(d_key,key,size,hipMemcpyHostToDevice);
    wrapper<<<1,1>>>(d_data,d_key);
    hipMemcpy(result, d_data, size, hipMemcpyDeviceToHost);
    for(int i=0;i<Nrow;i++){
	for(int j=0;j<Ncol;j++)
	    printf("%x\t",*(result+i*Nrow+j));
	printf("\n");
    }
    free(result);
    hipFree(d_data);
    hipFree(d_key);
    return 0;
}

__global__ void wrapper(uint8_t* block, uint8_t* rdk){
    addRoundKey(block,rdk);
    addRoundKey(block,rdk);
}
