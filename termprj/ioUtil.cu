#include "hip/hip_runtime.h"
#include "ioUtil.h"

off_t fsize(char* filename){
    struct stat st;
    if(stat(filename,&st)==0)
	return st.st_size;
    else 
	return -1;
}

__global__ void d_printx(void* data,size_t datasize){ 
    printf("on device:\n");
    for(int i=0;i<datasize;i++){
	printf("%x", *( (uint8_t*)data + i) );
    }
    printf("\n");
}
void printx(void* data,size_t datasize){ 
    printf("on host:\n");
    for(int i=0;i<datasize;i++){
	printf("%x",*((uint8_t*)data+i));
    }
    printf("\n");
}

