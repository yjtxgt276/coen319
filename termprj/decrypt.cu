#include "hip/hip_runtime.h"
#include "myAESlib.h"
#include "ioUtil.h"
#include "keySchedule.h"
#include "decrypt.h"
#include <string.h>
#include <sys/time.h>

int main(int argc, char** argv){
//timing
    struct timeval start,end;
    double exetime=0.0;
    gettimeofday(&start,NULL);
    if(argc!=3){
	printf("USAGE: %s [filename] [key]\n",argv[0]);
	return 1;
    }
    FILE* fp;
    if((fp=fopen(argv[1],"r"))==NULL){
	printf("ERR OPENING FILE %s.\n",argv[1]);
	return 1;
    }
    if(strlen(argv[2])>16){
	printf("The key should be less than 16 characters\n");
	return 1;
    }
    off_t filesize=fsize(argv[1]);
    printf("filesize: %lu bytes\n",filesize);

    char userkey[Nbyte];
    memset(userkey,0x00,Nbyte);

    strncpy(userkey,argv[2],strlen(argv[2]));
    printf("userkey:\n");
    printx(userkey,Nbyte);

    uint8_t* buffer = (uint8_t*) malloc(filesize);
    fread(buffer, filesize,1,fp);
    //printx(buffer,filesize);

    uint8_t roundkey[Nrd+1][Nrow][Ncol]; 
    memset(roundkey,0x00,(Nrd+1)*Nbyte);
    size_t rksize=(Nrd+1)*Nrow*Ncol;
    expandKey(userkey, roundkey);
    
    uint8_t *d_buffer,*d_rk;
    hipMalloc(&d_buffer,filesize);
    hipMemcpy(d_buffer,buffer,filesize,hipMemcpyHostToDevice);
    //d_printx<<<1,1>>>(d_buffer,filesize);

    hipMalloc(&d_rk,rksize);
    hipMemcpy(d_rk,roundkey,rksize,hipMemcpyHostToDevice);
    memset(buffer,0,filesize);

    //dim3 dimblock(1);
    //dim3 dimgrid(1);
    decrypt<<<GRIDSIZE,BLOCKSIZE>>>(d_buffer,d_rk,filesize);
    //d_printx<<<1,1>>>(d_buffer,filesize);

    hipMemcpy(buffer, d_buffer,filesize,hipMemcpyDeviceToHost);
    //printx(buffer,filesize);

    FILE* fpout;
    char outputname[256];
    strcpy(outputname,argv[1]);
    strcat(outputname,".plain");
    fpout = fopen(outputname,"w+");
    fwrite(buffer,filesize,1,fpout);
    free(buffer);
    //free(bufferout);
    hipFree(d_buffer);
    hipFree(d_rk);
    fclose(fp);
    fclose(fpout);

    gettimeofday(&end,NULL);
    exetime=(end.tv_sec*1000000+end.tv_usec-start.tv_sec*1000000-start.tv_usec)/1000000.0;
    printf("exe time: %f secs\n",exetime);
    return 0;
}

__global__ void decrypt(uint8_t* d_buffer, uint8_t* roundkey,off_t filesize){
    uint8_t *rdk, *dataptr;
    int thd = blockDim.x*blockIdx.x+threadIdx.x;
    int round =0;
    for(int pos=thd*Nbyte;pos<filesize;pos+=GRIDSIZE*BLOCKSIZE*Nbyte){
	    dataptr=d_buffer+pos; 	
	for(round=0;round<Nrd+1;round++){
	    rdk=roundkey+(Nrd-round)*Nbyte;
	    if(round == 10)
		addRoundKey(dataptr,rdk);
	    else if(round == 0){
		addRoundKey(dataptr,rdk);
		invShiftRow(dataptr);
		invByteSub(dataptr);
	    }
	    else {
		addRoundKey(dataptr,rdk);
		invMixCol(dataptr);
		invShiftRow(dataptr);
		invByteSub(dataptr);
		//printf("thd %d reading from byte %d:\n",thd,pos);
		//printf("\tinvmixcol %d\n",round);
	    }
	}
    }
    //printf("encrypting block %d,GPU\n",pos);
}
