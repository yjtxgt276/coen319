#include "hip/hip_runtime.h"
#include "byteSub.h"

__global__ void wrapper(uint8_t* block);

int main(){
    uint8_t data[Nrow*Ncol] = {0,1,2,3,4,5,6,7,8,9,10,11,12,13,}; 
    uint8_t* d_data;
    uint8_t* result;
    size_t size = Nrow*Ncol*sizeof(uint8_t);
    result =( uint8_t*) malloc(size);
    hipMalloc(&d_data, size);
    hipMemcpy(d_data,data, size, hipMemcpyHostToDevice);
    wrapper<<<1,1>>>(d_data);
    hipMemcpy(result, d_data,size,hipMemcpyDeviceToHost);
    for(int i=0;i<Nrow;i++){
	for(int j=0;j<Ncol;j++)
	    printf("%x\t",result[i*Ncol+j]);
	printf("\n");
    }
    free(result);
    hipFree(d_data);
    return 0;
}
__global__ void wrapper(uint8_t* block){
   byteSub(block); 
   invByteSub(block);
}
