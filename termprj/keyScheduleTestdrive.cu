#include "hip/hip_runtime.h"
#include "keySchedule.h"

int main(int argc, char** argv){
    char key[16];    
    memset(key,0x00,16);
    uint8_t roundkey[Nrd+1][Nrow][Ncol];
    /*
    roundkey = (uint8_t***) malloc(Nrd*sizeof(uint8_t**));
    for(int i=0;i<Nrd+1;i++){
	roundkey[i]=(uint8_t**) malloc(Nrow*sizeof(uint8_t*)); 
	for(int j=0;j<Nrow;j++){
	    roundkey[i][j]=(uint8_t*) malloc(Ncol*sizeof(uint8_t));
	}
    }

    for(int i=0;i<16;i++){
	key[i]=argv[1][i];
	//key[i]=0x00;
	printf("%x",key[i]);
    }
    */
    strncpy(key,argv[1],strlen(argv[1]));
    printf("key length: %d\n",strlen(argv[1]));
    printf("\n");
    expandKey(key,roundkey);

    for(int i=0;i<Nrd+1;i++){
	printf("round %d\n",i);
	for(int j=0;j<Nrow;j++){
	    for(int k=0;k<Ncol;k++){
		printf("%x\t",roundkey[i][j][k]);
	    }	
	    printf("\n");
	}
    }
    return 0;
}


