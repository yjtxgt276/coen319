#include "hip/hip_runtime.h"
#include "myAESlib.h"
#include "ioUtil.h"
#include "keySchedule.h"
#include "decrypt.h"
#include <string.h>

int main(int argc, char** argv){
    if(argc!=3){
	printf("USAGE: %s [filename] [key]\n",argv[0]);
	return 1;
    }
    FILE* fp;
    if((fp=fopen(argv[1],"r"))==NULL)
	printf("ERR OPENING FILE %s.\n",argv[1]);
    off_t filesize=fsize(argv[1]);
    printf("filesize=%lu\n",filesize);
    uint8_t* buffer = (uint8_t*) malloc(filesize);

    fread(buffer, filesize,1,fp);
    printx(buffer,filesize);
    uint8_t roundkey[Nrd+1][Nrow][Ncol]; 
    size_t rksize=(Nrd+1)*Nrow*Ncol;
    expandKey(argv[2], roundkey);
    
    uint8_t *d_buffer,*d_rk;
    hipMalloc(&d_buffer,filesize);
    hipMemcpy(d_buffer,buffer,filesize,hipMemcpyHostToDevice);
    d_printx<<<1,1>>>(d_buffer,filesize);
    hipMalloc(&d_rk,rksize);
    hipMemcpy(d_rk,roundkey,rksize,hipMemcpyHostToDevice);
    memset(buffer,0,filesize);

    //dim3 dimblock(1);
    //dim3 dimgrid(1);
    decrypt<<<GRIDSIZE,BLOCKSIZE>>>(d_buffer,d_rk,filesize);
    d_printx<<<1,1>>>(d_buffer,filesize);
    hipMemcpy(buffer, d_buffer,filesize,hipMemcpyDeviceToHost);
    printx(buffer,filesize);

    FILE* fpout;
    fpout = fopen(argv[2],"w+");
    fwrite(buffer,filesize,1,fpout);
    free(buffer);
    //free(bufferout);
    hipFree(d_buffer);
    hipFree(d_rk);
    fclose(fp);
    fclose(fpout);
    return 0;
}

__global__ void decrypt(uint8_t* d_buffer, uint8_t* roundkey,off_t filesize){
    uint8_t *rdk, *dataptr;
    int thd = blockDim.x*blockIdx.x+threadIdx.x;
    int round =0;
    for(int pos=thd*Nbyte;pos<filesize;pos+=GRIDSIZE*BLOCKSIZE*Nbyte){
	    dataptr=d_buffer+pos; 	
	for(round=0;round<Nrd+1;round++){
	    rdk=roundkey+(Nrd-round)*Nbyte;
	    if(round == 10)
		addRoundKey(dataptr,rdk);
	    else if(round == 0){
		addRoundKey(dataptr,rdk);
		invShiftRow(dataptr);
		invByteSub(dataptr);
	    }
	    else {
		addRoundKey(dataptr,rdk);
		invMixCol(dataptr);
		invShiftRow(dataptr);
		invByteSub(dataptr);
		printf("thd %d reading from byte %d:\n",thd,pos);
		printf("\tinvmixcol %d\n",round);
	    }
	}
    }
    //printf("encrypting block %d,GPU\n",pos);
}
