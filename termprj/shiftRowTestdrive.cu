#include "hip/hip_runtime.h"
#include "shiftRow.h"

__global__ void wrapper(uint8_t* block);
int main(){
// try copy 2d to dev using cudamemcpy, works
    uint8_t data[Nrow][Ncol] = {0,1,2,3,0,1,2,3,0,1,2,3,0,1,2,3}; 
    uint8_t* d_data;
    uint8_t* result;
    size_t size = Nrow*Ncol*sizeof(uint8_t);
    result =( uint8_t*) malloc(size);
    hipMalloc(&d_data, size);
    hipMemcpy(d_data,data,size,hipMemcpyHostToDevice);
    wrapper<<<1,1>>>(d_data);
    hipMemcpy(result, d_data, size, hipMemcpyDeviceToHost);
    for(int i=0;i<Nrow;i++){
	for(int j=0;j<Ncol;j++)
	    printf("%x\t",*(result+i*Nrow+j));
	printf("\n");
    }
    free(result);
    hipFree(d_data);
    return 0;
}
__global__ void wrapper(uint8_t* block){
   shiftRow(block); 
   invShiftRow(block);
}
