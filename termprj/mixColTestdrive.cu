#include "hip/hip_runtime.h"
#include "mixCol.h"

__global__ void wrapper(uint8_t* block);
int main(){
    uint8_t data[][Ncol]={0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15}; 
    uint8_t* d_data;
    size_t size = Ncol*Nrow*sizeof(uint8_t);
    hipMalloc(&d_data, size);
    hipMemcpy(d_data,data,size, hipMemcpyHostToDevice);
    wrapper<<<1,1>>>(d_data);
    //for(int i=0)
    uint8_t* result =( uint8_t*) malloc(size);
    hipMemcpy(result, d_data, size, hipMemcpyDeviceToHost);
    for(int i=0;i<Nrow;i++){
	for(int j=0;j<Ncol;j++)
	    printf("%x\t",*(result+i*Nrow+j));
	printf("\n");
    }
    hipFree(d_data);
    return 0;
}
__global__ void wrapper(uint8_t* block){
   mixCol(block); 
   invMixCol(block);
}
