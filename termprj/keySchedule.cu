#include "hip/hip_runtime.h"
#include "keySchedule.h"

void transform(uint8_t rki[Nrow][Ncol], uint8_t* temp, int round){
//round constant
    uint8_t rc=roundconst[round];
//shift the col up by 1 ele, and sub with sbox
    for(int i=0;i<Nrow;i++)
	temp[i]=sbox[ rki[((i+1)%Nrow)][Ncol-1] ];
    temp[0] = temp[0] ^ rc;
    return ;
}

void expandKey(void* k, uint8_t rk[Nrd+1][Nrow][Ncol])
{
    uint8_t* kp = (uint8_t*) k;
    uint8_t key[Nrow][Ncol]; 
//pad the key
    uint8_t temp[Nrow];
//input key -> matrixed key
    for(int i=0;i<Nrow;i++)
	for(int j=0;j<Ncol;j++){
	    key[i][j] = kp[i*Nrow+j]; 
	    //printf("%x\t",key[i][j]);
	}
//expand the key
    for(int round=0;round<Nrd+1;round++)
    {
//round 0
	if(round==0)
	    for(int row=0;row<Nrow;row++)
		for(int col=0;col<Ncol;col++)
		    rk[round][row][col]=key[row][col];    	
// other rounds
	else
	{
	    for(int col=0;col<Ncol;col++)
	    {
//first col of each roundkey
		if(col==0)
		{
		    transform(rk[round-1],temp,round);
		    for(int row=0;row<Nrow;row++)
			rk[round][row][col]=rk[round-1][row][col] ^ temp[row]; 
		}
//other cols of each roundkey
		else
		    for(int row=0;row<Nrow;row++)
			rk[round][row][col]=rk[round-1][row][col] ^ rk[round][row][col-1]; 
	    }	
	}
    }
}
