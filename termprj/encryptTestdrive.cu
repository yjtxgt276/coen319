#include "hip/hip_runtime.h"
#include "myAESlib.h"
#include "ioUtil.h"
#include "keySchedule.h"
#include "encrypt.h"
#include <string.h>

int main(int argc, char** argv){
    if(argc!=3){
	printf("USAGE: %s [filename] [key]\n",argv[0]);
	return 1;
    }
    FILE* fp;
    if((fp=fopen(argv[1],"r"))==NULL)
	printf("ERR OPENING FILE %s.\n",argv[1]);
    off_t filesize=fsize(argv[1]);
    printf("filesize=%lu\n",filesize);
    uint8_t* buffer = (uint8_t*) malloc(filesize);

    fread(buffer,sizeof(uint8_t),filesize,fp);
    printx(buffer,filesize);
    uint8_t roundkey[Nrd+1][Nrow][Ncol]; 
    size_t rksize=(Nrd+1)*Nrow*Ncol;
    expandKey(argv[2], roundkey);

    uint8_t *d_buffer,*d_rk;
    hipMalloc(&d_buffer,filesize);
    hipMemcpy(d_buffer,buffer,filesize,hipMemcpyHostToDevice);
    d_printx<<<1,1>>>(d_buffer,filesize);
    hipMalloc(&d_rk,rksize);
    hipMemcpy(d_rk,roundkey,rksize,hipMemcpyHostToDevice);
    memset(buffer,0,filesize);

    encrypt<<<GRIDSIZE,BLOCKSIZE>>>(d_buffer,d_rk,filesize);
    d_printx<<<1,1>>>(d_buffer,filesize);
    hipMemcpy(buffer, d_buffer,filesize,hipMemcpyDeviceToHost);
    printx(buffer,filesize);

    FILE* fpout;
    fpout = fopen(argv[2],"w+");
    fwrite(buffer,filesize,1,fpout);
    free(buffer);

    hipFree(d_buffer);
    hipFree(d_rk);
    fclose(fp);
    fclose(fpout);
    return 0;
}

__global__ void encrypt(uint8_t* d_buffer, uint8_t* roundkey,off_t filesize){
    uint8_t *rdk, *dataptr;
    int thd = blockDim.x*blockIdx.x+threadIdx.x;
    int round=0;
    for(int pos=thd*Nbyte;pos<filesize;pos+=GRIDSIZE*BLOCKSIZE*Nbyte){
	    dataptr=d_buffer+pos; 	
	for(round=0;round<Nrd+1;round++){
	    rdk=roundkey+Nbyte*round;
	    if(round == 0)
		addRoundKey(dataptr,rdk);
	    else if(round == 10){
		byteSub(dataptr);
		shiftRow(dataptr);
		addRoundKey(dataptr,rdk);
	    }
	    else {
		byteSub(dataptr);
		shiftRow(dataptr);
		mixCol(dataptr);
		addRoundKey(dataptr,rdk);
		printf("thd %d reading from byte %d:\n",thd,pos);
		printf("\tmixcol %d\n",round);
	    }
	}
    }
    //printf("encrypting block %d,GPU\n",pos);
}
